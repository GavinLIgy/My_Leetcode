#include "hip/hip_runtime.h"
template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_sum(float val){
    #pragma unroll
    for (int mask = kWarpSize >> 1; mask >= 1; mask >>= 1){
        val += __shfl_xor_sync(0xffffffff, val, mask);
    }
    return val;
}

template<const int kWarpSize = WARP_SIZE>
__device__ __forceinline__ float warp_reduce_max(float val){
    #pragma unroll
    for(int mask = kWarpSize >> 1; mask >= 1; mask >>= 1){
        val = fmaxf(val, __shfl_xor_sync(0xffffffff, val, mask));
    }
    return val;
}
