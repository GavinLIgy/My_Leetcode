#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

// naive imple, compute one c[i,j] element per threads, all row major
__global__ void sgemm_naive_f32_kernel(float* a, float* b, float* c, int M, int N, int K){
    int n = blockIdx.x * blockDim.x + threadIdx.x ;
    int m = blockIdx.y * blockDim.y + threadIdx.y ;

    if(n < N && m < M) {
        float psum = 0.0f;
        #pragma unroll
        for(int k=0; k<K; k++){
            psum += a[m*K + k] * b[k*N + n];
        }
        c[m*N + n] = psum;
    }
}

// block tile + K tile with smem
// block tile (BM, BN) + K tile (BK=32)
// gird((N + BN -1) / BN, (M + BM -1) / BM), block(BN, BM)
// a: M x K, b: K x N, c: M x N, compute: c = a*b, all row major
template<const int BM = 32, const int BN = 32, const int BK = 32>
__global__ void sgemm_sliced_k_f32_kernel(float* a, float* b, float* c, int M, int N, int K){
    // [1] Block tile: using 32 x 32 block for c element
    // [2] k tile: using shared memory, tile k elements into bk-elements tiles
    // constexpr int BM = 32;
    // constexpr int BN = 32;
    // constexpr int BK = 32;
    __shared__ float s_a[BM][BK], s_b[BK][BN];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tid = threadIdx.y * blockDim.x + tx; // tid within the block
    // load values to shared memory, 32x32 threads working together 
    // to fetch data along the row direction of a and b both for s_a 
    // and s_b 32x32x4x2=8KB, we use 32x32 threads within block to 
    // load 32x32 elements from global memory to shared memory, namely, 
    // each thread will load 1 element.
    int load_smem_a_m = tid / 32; // tid / BM, threadIdx.y
    int load_smem_a_k = tid % 32; // tid % BK, threadIdx.x
    int load_smem_b_k = tid / 32; // tid / BK, threadIdx.y
    int load_smem_b_n = tid % 32; // tid % BN, threadIdx.x
    int load_gmem_a_m = by * BM + load_smem_a_m; // global row of a and c
    int load_gmem_b_n = bx * BN + load_smem_b_n; // global col of b and c
    if (load_gmem_a_m >= M || load_gmem_b_n >= N) return;

    float sum = 0.f;
    for(int bk=0; bk < (K + BK - 1) / BK; ++bk){
        int load_gemm_a_k = bk * BK + load_smem_a_k;
        int load_gemm_a_addr = load_gmem_a_m * K + load_gemm_a_k;// row major
        s_a[load_smem_a_m][load_smem_a_k] = a[load_gemm_a_addr];
        int load_gemm_b_k = bk * BK + load_smem_b_k;
        int load_gemm_b_addr = load_gemm_b_k * N + load_gemm_b_n;
        s_a[load_smem_b_k][load_smem_b_n] = a[load_gemm_b_addr];
        __syncthreads();
        #pragma unroll
        for(int k=0; k<BK; ++k){
            int comp_smem_a_m = load_smem_a_m;
            int comp_smem_b_n = load_smem_b_n;
            sum += s_a[comp_smem_a_m][k] * s_b[k][comp_smem_b_n];
        }
        __syncthreads();
    }
    int store_gemm_c_m = load_gmem_a_m;
    int store_gemm_c_n = load_gmem_b_n;
    int store_genn_c_addr = store_gemm_c_m * N + store_gemm_c_n;
    c[store_genn_c_addr] = sum;
}
